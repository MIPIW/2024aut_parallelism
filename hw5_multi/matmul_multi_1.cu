#include "hip/hip_runtime.h"
#include "matmul_multi.h"
#include "util.h"
#include <pthread.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#define MAX_NUM_GPU 4
#define TILE_SIZE 16
int num_devices = 0;

typedef struct {
    int device_id;
    const float *A;
    const float *B;
    float *C;
    int M, N, K;
    int Mbegin, Mend;
} ThreadData;


// __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
//                               int K) {
//   int i = blockDim.x * blockIdx.x + threadIdx.x;
//   int j = blockDim.y * blockIdx.y + threadIdx.y;
//   if (i >= M || j >= N)
//     return;

//   C[i * N + j] = 0;
//   for (int k = 0; k < K; ++k) {
//     C[i * N + j] += A[i * K + k] * B[k * N + j];
//   }
// }


// __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
//                               int K) {
//     int row = blockIdx.x;
//     int col = blockIdx.y;

//     // Ensure we are within bounds
//     if (row >= M || col >= N) return;

//     // Compute the dot product of row from A and column from B
//     float sum = 0.0f;
//     for (int k = 0; k < K; k++) {
//         sum += A[row * K + k] * B[k * N + col];
//     }

//     // Write the computed value to the output matrix
//     C[row * N + col] = sum;
// }

__global__ void matmul_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    // Shared memory for storing tiles of A and B
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Calculate thread row and column within the output matrix
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Accumulator for the dot product
    float sum = 0.0f;

    // Loop over tiles of A and B required for C[row, col]
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load elements into shared memory (if within bounds)
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            tileA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K) {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Synchronize to ensure all threads have loaded their tiles
        __syncthreads();

        // Compute partial dot product for the current tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        // Synchronize before loading the next tile
        __syncthreads();
    }

    // Write the result to the output matrix (if within bounds)
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Array of device (GPU) pointers
// static float *a_d[MAX_NUM_GPU];
// static float *b_d[MAX_NUM_GPU];
// static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];

void *threaded_matmul(void *arg) {
    ThreadData *data = (ThreadData *)arg;

    // Set GPU for this thread
    CUDA_CALL(hipSetDevice(data->device_id));

    // Allocate memory on the GPU
    float *a_d, *b_d, *c_d;
    CUDA_CALL(hipMalloc(&a_d, (data->Mend - data->Mbegin) * data->K * sizeof(float)));
    CUDA_CALL(hipMalloc(&b_d, data->K * data->N * sizeof(float)));
    CUDA_CALL(hipMalloc(&c_d, (data->Mend - data->Mbegin) * data->N * sizeof(float)));

    // Copy matrices to the GPU
    CUDA_CALL(hipMemcpy(a_d, data->A + data->Mbegin * data->K,
                         (data->Mend - data->Mbegin) * data->K * sizeof(float),
                         hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(b_d, data->B, data->K * data->N * sizeof(float),
                         hipMemcpyHostToDevice));

    // Configure grid and block dimensions
    dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
    dim3 gridDim((data->N + TILE_SIZE - 1) / TILE_SIZE,
                 (data->Mend - data->Mbegin + TILE_SIZE - 1) / TILE_SIZE,
                 1);

    // Launch the kernel
    matmul_kernel<<<gridDim, blockDim>>>(a_d, b_d, c_d, data->M, data->N, data->K);

    // Synchronize GPU
    CUDA_CALL(hipDeviceSynchronize());

    // Copy the result matrix back to host memory
    CUDA_CALL(hipMemcpy(data->C + data->Mbegin * data->N, c_d,
                         (data->Mend - data->Mbegin) * data->N * sizeof(float),
                         hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CALL(hipFree(a_d));
    CUDA_CALL(hipFree(b_d));
    CUDA_CALL(hipFree(c_d));

    return NULL;
}

// void matmul(const float *A, const float *B, float *C, int M, int N, int K) {

//   // Upload A and B matrix to every GPU
//   for (int i = 0; i < num_devices; i++) {
//     CUDA_CALL(hipMemcpy(a_d[i], A + Mbegin[i] * K,
//                          (Mend[i] - Mbegin[i]) * K * sizeof(float),
//                          hipMemcpyHostToDevice));
//     CUDA_CALL(hipMemcpy(b_d[i], B, K * N * sizeof(float), hipMemcpyHostToDevice));
//   }

//   // Launch kernel on every GPU
//   for (int i = 0; i < num_devices; i++) {
//     // dim3 blockDim(1, 1, 1);
//     // dim3 gridDim(Mend[i] - Mbegin[i], N, 1);

//     dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
//     dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (Mend[i] - Mbegin[i] + TILE_SIZE - 1) / TILE_SIZE, 1);

//     CUDA_CALL(hipSetDevice(i));
//     matmul_kernel<<<gridDim, blockDim>>>(a_d[i], b_d[i], c_d[i], M, N, K);
//   }

//   for (int i = 0; i < num_devices; i++) {
//     CUDA_CALL(hipDeviceSynchronize());
//   }

//   // Download C matrix from GPUs
//   for (int i = 0; i < num_devices; i++) {
//     CUDA_CALL(hipMemcpy(C + Mbegin[i] * N, c_d[i],
//                          (Mend[i] - Mbegin[i]) * N * sizeof(float),
//                          hipMemcpyDeviceToHost));
//   }

//   // DO NOT REMOVE; NEEDED FOR TIME MEASURE
//   for (int i = 0; i < num_devices; i++) {
//     CUDA_CALL(hipDeviceSynchronize());
//   }
// }

void * temp(void *arg){
  return NULL;
}

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {
    pthread_t threads[MAX_NUM_GPU];
    ThreadData thread_data[MAX_NUM_GPU];

    // Create and launch threads
    for (int i = 0; i < num_devices; i++) {
        thread_data[i].device_id = i;
        thread_data[i].A = A;
        thread_data[i].B = B;
        thread_data[i].C = C;
        thread_data[i].M = M;
        thread_data[i].N = N;
        thread_data[i].K = K;
        thread_data[i].Mbegin = Mbegin[i];
        thread_data[i].Mend = Mend[i];

        pthread_create(&threads[i], NULL, threaded_matmul, &thread_data[i]);
        // pthread_create(&threads[i], NULL, temp, &thread_data[i]);

    }

    // Join threads
    for (int i = 0; i < num_devices; i++) {
        pthread_join(threads[i], NULL);
    }
}


void matmul_initialize(int M, int N, int K) {

  CUDA_CALL(hipGetDeviceCount(&num_devices));

  printf("Using %d devices\n", num_devices);
  for (int i = 0; i < num_devices; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // Try printing more detailed information here
    printf("GPU %d: %s\n", i, prop.name);
  }

  if (num_devices <= 0) {
    printf("No CUDA device found. Aborting\n");
    exit(1);
  }

  // Setup problem size for each GPU
  for (int i = 0; i < num_devices; i++) {
    Mbegin[i] = (M / num_devices) * i;
    Mend[i] = (M / num_devices) * (i + 1);
  }
  Mend[num_devices - 1] = M;

  // Allocate device memory for each GPU
  // for (int i = 0; i < num_devices; i++) {
  //   CUDA_CALL(hipSetDevice(i));
  //   CUDA_CALL(hipMalloc(&a_d[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
  //   CUDA_CALL(hipMalloc(&b_d[i], K * N * sizeof(float)));
  //   CUDA_CALL(hipMalloc(&c_d[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  // }
}

void matmul_finalize() {

  // Free all GPU memory
  // for (int i = 0; i < num_devices; i++) {
  //   CUDA_CALL(hipFree(a_d[i]));
  //   CUDA_CALL(hipFree(b_d[i]));
  //   CUDA_CALL(hipFree(c_d[i]));
  // }
}
