#include "hip/hip_runtime.h"
#include "matmul_single.h"
#include "util.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#define TILE_SIZE 16
#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

// __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
//                               int K) {
//   int i = blockDim.x * blockIdx.x + threadIdx.x;
//   int j = blockDim.y * blockIdx.y + threadIdx.y;
//   if (i >= M || j >= N)
//     return;

//   C[i * N + j] = 0;
//   for (int k = 0; k < K; ++k) {
//     C[i * N + j] += A[i * K + k] * B[k * N + j];
//   }
// }


__global__ void matmul_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    // Shared memory for storing tiles of A and B
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Calculate thread row and column within the output matrix
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Accumulator for the dot product
    float sum = 0.0f;

    // Loop over tiles of A and B required for C[row, col]
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load elements into shared memory (if within bounds)
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            tileA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K) {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Synchronize to ensure all threads have loaded their tiles
        __syncthreads();

        // Compute partial dot product for the current tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        // Synchronize before loading the next tile
        __syncthreads();
    }

    // Write the result to the output matrix (if within bounds)
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}



// Array of device (GPU) pointers
static float *a_d;
static float *b_d;
static float *c_d;

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {

  // Upload A and B matrix to every GPU
  CUDA_CALL(hipMemcpy(a_d, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(b_d, B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // Launch kernel on every GPU
  dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
  dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, 1);

  matmul_kernel<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K);

  CUDA_CALL(hipDeviceSynchronize());

  // Download C matrix from GPUs
  CUDA_CALL(hipMemcpy(C, c_d, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CUDA_CALL(hipDeviceSynchronize());
}

void matmul_initialize(int M, int N, int K) {
  
  int num_devices;
  // Only root process do something
  CUDA_CALL(hipGetDeviceCount(&num_devices));

  if (num_devices <= 0) {
    printf("No CUDA device found. Aborting\n");
    exit(1);
  }

  // Allocate device memory 
  CUDA_CALL(hipMalloc(&a_d, M * K * sizeof(float)));
  CUDA_CALL(hipMalloc(&b_d, K * N * sizeof(float)));
  CUDA_CALL(hipMalloc(&c_d, M * N * sizeof(float)));
}

void matmul_finalize() {

  // Free GPU memory
  CUDA_CALL(hipFree(a_d));
  CUDA_CALL(hipFree(b_d));
  CUDA_CALL(hipFree(c_d));
}
